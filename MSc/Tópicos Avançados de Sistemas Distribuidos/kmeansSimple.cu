
#include <hip/hip_runtime.h>
#include <vector>
#include <random>
#include <limits>
#include <iostream>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <chrono>

#define ITERATIONS 200

struct Point
{
	float x{0}, y{0};
};

using DataFrame = std::vector<Point>;

__global__ void cudaClear(Point *points, const int bytes)
{
	const int index = threadIdx.x;

	if(index < bytes)
		points[index] = {0,0};

	__syncthreads();

}

struct CUDADataFrame
{
	CUDADataFrame(int size): size(size), bytes(size * sizeof(Point))
	{
		hipMalloc((void **)&points, bytes);
	}

	CUDADataFrame(int size, DataFrame df): size(size), bytes(size * sizeof(Point))
	{
		hipMalloc((void **)&points, bytes);
		hipMemcpy(points, df.data(), bytes, hipMemcpyHostToDevice);
	}

	~CUDADataFrame()
	{
		hipFree(points);
	}

	void clear()
	{
		cudaClear<<<1, size>>>(points, bytes);
	}

	Point *points;
	int size{0};
	int bytes{0};
};

__device__ float square(float val)
{
	return val * val;
}

__device__ float squared_distance(Point first, Point second)
{
	return square(first.x - second.x) + square(first.y - second.y);
}

__global__ void assign_clusters(const Point *data_set, int data_size, const Point *means, Point *new_sums, int k, int *counts)
{
	const int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(index >= data_size) return;

	const Point point = data_set[index];

	float best_distance = 999999;
	int best_cluster = 0;
	for(int cluster = 0; cluster < k; cluster++)
	{
		const float distance = squared_distance(point, means[cluster]);

		if(distance < best_distance)
		{
			best_distance = distance;
			best_cluster = cluster;
		}
	}

	atomicAdd(&new_sums[best_cluster].x, point.x);
	atomicAdd(&new_sums[best_cluster].y, point.y);
	atomicAdd(&counts[best_cluster], 1);
}

__global__ void compute_new_means(Point *means, const Point *new_sums, const int *counts)
{
	const int cluster = threadIdx.x;
	const int count = max(1, counts[cluster]);
	means[cluster].x = new_sums[cluster].x / count;
	means[cluster].y = new_sums[cluster].y / count;
}

void place_random_centroids(DataFrame &data)
{
	std::mt19937 rng(std::random_device{}());
	std::shuffle(data.begin(), data.end(), rng);
}

int main(int argc, char const *argv[])
{
	if(argc < 3)
	{
		std::cerr << "usage: assign_clusters <data-file> <k> [iterations]" << std::endl;
		std::exit(EXIT_FAILURE);
	}

	const auto k = std::atoi(argv[2]);
	const auto n_of_iterations = (argc == 4) ? std::atoi(argv[3]) : ITERATIONS;

	DataFrame df;

	std::ifstream data_file(argv[1]);
    if(!data_file)
    {
        std::cerr << "Could not open file: " << argv[1] << std::endl;
        std::exit(EXIT_FAILURE);
    }

    std::string line;
    while (std::getline(data_file, line)) {
        std::istringstream line_stream(line);	
        Point point;
        uint16_t label;
        line_stream >> point.x >> point.y >> label;
        df.push_back(point);
    }   

    const int n_of_elements = df.size();

    CUDADataFrame device_data(n_of_elements, df);

    place_random_centroids(df);

    CUDADataFrame device_means(k, df);
    CUDADataFrame device_sums(k);

    int *device_counts;
    hipMalloc(&device_counts, k * sizeof(int));
    hipMemset(device_counts, 0, k * sizeof(int));

    const int threads = 1024;
    const int blocks = (n_of_elements + threads - 1) / threads;


    const auto start = std::chrono::high_resolution_clock::now();
    for(int iter = 0; iter < n_of_iterations; iter++)
    {
    	hipMemset(device_counts, 0, k * sizeof(int));

    	device_sums.clear();

    	assign_clusters<<<blocks, threads>>>(device_data.points, n_of_elements, device_means.points, device_sums.points, k, device_counts);

    	hipDeviceSynchronize();

    	compute_new_means<<<1, k>>>(device_means.points, device_sums.points, device_counts);

    	hipDeviceSynchronize();
    }
  	const auto end = std::chrono::high_resolution_clock::now();
  	const auto duration = std::chrono::duration_cast<std::chrono::duration<float>>(end - start);
  	std::cerr << "Time: " << duration.count() << "s" << std::endl;

    hipFree(device_counts);

    DataFrame result(k);
    hipMemcpy(result.data(), device_means.points, device_means.bytes, hipMemcpyDeviceToHost);

    for(int cluster = 0; cluster < k; cluster++)
    {
    	std::cout << result[cluster].x << " " << result[cluster].y << std::endl;
    }
    
	return 0;
}
