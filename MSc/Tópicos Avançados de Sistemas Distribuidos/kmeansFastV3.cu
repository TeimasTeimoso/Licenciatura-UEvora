
#include <hip/hip_runtime.h>
#include <vector>
#include <random>
#include <limits>
#include <iostream>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <chrono>

#define ITERATIONS 200

struct Point
{
    float x{0}, y{0};
};

using DataFrame = std::vector<Point>;

__global__ void cudaClear(Point *points, const int bytes)
{
    const int index = threadIdx.x;

    if(index < bytes)
        points[index] = {0,0};

    __syncthreads();

}

struct CUDADataFrame
{
    CUDADataFrame(int size): size(size), bytes(size * sizeof(Point))
    {
        hipMalloc((void **)&points, bytes);
    }

    CUDADataFrame(int size, DataFrame df): size(size), bytes(size * sizeof(Point))
    {
        hipMalloc((void **)&points, bytes);
        hipMemcpy(points, df.data(), bytes, hipMemcpyHostToDevice);
    }

    ~CUDADataFrame()
    {
        hipFree(points);
    }

    void clear()
    {
        cudaClear<<<1, size>>>(points, bytes);
    }

    Point *points;
    int size{0};
    int bytes{0};
};

__device__ float square(float val)
{
    return val * val;
}

__device__ float squared_distance(Point first, Point second)
{
    return square(first.x - second.x) + square(first.y - second.y);
}

__global__ void assign_clusters(const Point *data_set, int set_size, const Point *means, Point *new_sums, int k, int *counts)
{

    extern __shared__ float shared_data[];


    // Index of thread in it's own block
    const int local_index = threadIdx.x;
    const int global_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Set all positions to 0 TEST
    //shared_data[global_index] = 0;

    if(global_index >= set_size) return;


    // Load the means value into shared memory.
    if(local_index < k)
    {
        shared_data[local_index] = means[local_index].x;
        shared_data[local_index + k] = means[local_index].y;
    }

    __syncthreads();

    const Point data_point = data_set[global_index];

    float best_distance = 99999999;
    int best_cluster = -1;

    for(int cluster = 0; cluster < k; cluster++)
    {
        Point p = {shared_data[cluster], shared_data[cluster + k]};
 
        float distance = squared_distance(data_point, p);

        if(distance < best_distance)
        {
            best_distance = distance;
            best_cluster = cluster;
        }
    }

    __syncthreads();

    const int x = local_index;
    const int y = local_index + blockDim.x;
    const int count = local_index + blockDim.x + blockDim.x;

    for(int cluster = 0; cluster < k; cluster++)
    {
        shared_data[x] = (best_cluster == cluster) ? data_point.x : 0;
        shared_data[y] = (best_cluster == cluster) ? data_point.y : 0;
        shared_data[count] = (best_cluster == cluster) ? 1 : 0;

        __syncthreads();

        // reduction for this cluster
        for(int stride = blockDim.x / 2; stride > 0; stride >>= 1)
        {
            if(local_index < stride)
            {
                shared_data[x] += shared_data[x + stride];
                shared_data[y] += shared_data[y + stride];
                shared_data[count] += shared_data[count + stride];
            }
            __syncthreads();
        }

        if(local_index == 0)
        {
            const int cluster_index = blockIdx.x * k + cluster;
            new_sums[cluster_index].x = shared_data[x];
            new_sums[cluster_index].y = shared_data[y];
            counts[cluster_index] = shared_data[count];
        }
        __syncthreads();
    }
}


__global__ void compute_new_means(Point *means, Point *new_sums, int k, int *counts)
{
    extern __shared__ float shared_data[];

    const int index = threadIdx.x;
    const int y_offset = blockDim.x;

    shared_data[index] = new_sums[index].x;
    shared_data[index + y_offset] = new_sums[index].y;
    __syncthreads();

    for(int stride = blockDim.x / 2; stride >= k; stride >>= 1)
    {
        if(index < stride)
        {
            shared_data[index] += shared_data[index + stride];
            shared_data[index + y_offset] += shared_data[index + y_offset + stride];
        }
        __syncthreads();
    }



    if(index < k)
    {
        const int count = max(1, counts[index]);
        means[index].x = shared_data[index] / count;
        means[index].y = shared_data[index + y_offset] / count;
        new_sums[index] = {0,0};
    	counts[index] = 0;
    }
}

void place_random_centroids(DataFrame &data)
{
    std::mt19937 rng(std::random_device{}());
    std::shuffle(data.begin(), data.end(), rng);
}

int main(int argc, char const *argv[])
{
    if(argc < 3)
    {
        std::cerr << "usage: assign_clusters <data-file> <k> [iterations]" << std::endl;
        std::exit(EXIT_FAILURE);
    }

    const auto k = std::atoi(argv[2]);
    const auto n_of_iterations = (argc == 4) ? std::atoi(argv[3]) : ITERATIONS;

    DataFrame df;

    std::ifstream data_file(argv[1]);
    if(!data_file)
    {
        std::cerr << "Could not open file: " << argv[1] << std::endl;
        std::exit(EXIT_FAILURE);
    }

    std::string line;
    while (std::getline(data_file, line)) {
        std::istringstream line_stream(line);   
        Point point;
        int label;
        line_stream >> point.x >> point.y >> label;
        df.push_back(point);
    }   

    const int n_of_elements = df.size();

    CUDADataFrame device_data(n_of_elements, df);

    place_random_centroids(df);

    CUDADataFrame device_means(k, df);

    const int threads = 1024;
    const int blocks = (n_of_elements + threads - 1) / threads;

    const int fine_shared_memory = 3 * threads * sizeof(float);
    const int coarse_shared_memory = 2 * k * blocks * sizeof(float);

    CUDADataFrame device_sums(k * blocks);
    int *device_counts;
    hipMalloc(&device_counts, k * blocks * sizeof(int));
    hipMemset(device_counts, 0, k * blocks * sizeof(int));

    const auto start = std::chrono::high_resolution_clock::now();
    for(int iter = 0; iter < n_of_iterations; iter++)
    {
        assign_clusters<<<blocks, threads, fine_shared_memory>>>(device_data.points, n_of_elements, device_means.points, device_sums.points, k, device_counts);

        hipDeviceSynchronize();

        compute_new_means<<<1, k * blocks, coarse_shared_memory>>>(device_means.points, device_sums.points, k, device_counts);

        hipDeviceSynchronize();
    }
  	const auto end = std::chrono::high_resolution_clock::now();
  	const auto duration = std::chrono::duration_cast<std::chrono::duration<float>>(end - start);
  	std::cerr << "Time: " << duration.count() << "s" << std::endl;

    hipFree(device_counts);

    DataFrame result(k);
    hipMemcpy(result.data(), device_means.points, device_means.bytes, hipMemcpyDeviceToHost);

    for(int cluster = 0; cluster < k; cluster++)
    {
        std::cout << result[cluster].x << " " << result[cluster].y << std::endl;
    }


    return 0;
}
